#include "hip/hip_runtime.h"

//#define US_SIGNS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
//#include <cutil.h>
#include "ssd_fft_gpu_kernel.cu"
//#define BUILD_DLL
#include <GL/glew.h>
#include <GL/glut.h>
//#include "include/ssd_fft_gpu_dll.h"
#include <ssd_fft_gpu_common.h>
#include "include/ssd_fft_gpu.h"
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>
#include <helper_timer.h>

#define CUTFalse false
#define CUTTrue true
#define CUTBoolean bool
#define CUDA_SAFE_CALL checkCudaErrors
#define CUFFT_SAFE_CALL checkCudaErrors
#define CUT_SAFE_CALL checkCudaErrors
#define CUT_CHECK_ERROR getLastCudaError
#define cutComparefe sdkCompareL2fe
#define cutCreateTimer sdkCreateTimer


extern "C"
int CLAHE(unsigned char* pImage, unsigned int uiXRes, unsigned int uiYRes, unsigned char Min,
	unsigned char Max, unsigned int uiNrX, unsigned int uiNrY,
	unsigned int uiNrBins, float fCliplimit);

#define gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx)  gd_afCompFlt + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) +  ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
#define d_pafWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx) d_pafWholeTplFFT + ((giScnH * giScnW * giNumIPInFirst * giNumSz) * iFltAbsIndx) + ((giScnH * giScnW * giNumIPInFirst) * iSzIndx) + ((giScnH * giScnW) * iIPAbsIndx)
#define gd_afWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx) gd_afWholeTplFFT + ((giScnH * giScnW * giNumIPInFirst * giNumSz) * iFltAbsIndx) + ((giScnH * giScnW * giNumIPInFirst) * iSzIndx) + ((giScnH * giScnW) * iIPAbsIndx)
#define d_pafPartTplFFT(iIPIndx, iSzIndx, iFltIndx) d_pafPartTplFFT + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) + ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
#define gd_afPartTplFFT(iIPIndx, iSzIndx, iFltIndx) gd_afPartTplFFT + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) + ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
////////////////////////////////////////////////////////////////////////////////
// Global vars
////////////////////////////////////////////////////////////////////////////////
size_t wgSize[3] = { 1, 1, 1 };
size_t gSize[3] = { 1, 1, 1 };
//trashold for the PSR (might be different for day and night)
//const float gfPSRTrashold = 8.0f;
const float gfPSRTrashold = 7.5f;
//params related to Majority Voting
//keep track of PSRs for giTrackingLen frames
const float giTrackingLen = 10;
float giFrameNo = 0;
int giNumFramesInAcc = 0; //number of frames that contribute to AccPSR
						  //max acc psr should be greater than gfAccPSRTrashold so that we can conclude that speed sign is recognized
float gfAccPSRTrashold = 0;
//factor which determines additional confidence due to IP (if IP is equal to prevIP increase conf). 
//makes sense when different IP Rots are defined.
const float gfAddConfIPFac = 0.25;
//factor which determines additional confidence due to Sz (if Sz is larger to prevSz increase conf). 
const float gfAddConfEqSzFac = 0.5;
const float gfAddConfGrSzFac = 1.25;

typedef struct AccRes_struct
{
	float fAccConf;
	int iPrevIP;
	int iPrevSz;
}AccRes_struct_t;

AccRes_struct_t* gastAccRes;


//for PSR calculation define sidelobe
//area = frame+mask
const int giAreaH = 20;
const int giMaskH = 4;

const int	giScnSz = giScnW * giScnH;
const int	giScnMemSzReal = giScnSz * sizeof(hipfftReal);
const int   giScnMemSzCmplx = giScnSz * sizeof(hipfftComplex);
const int   giScnMemSzUChar = giScnSz * sizeof(unsigned char);
const int	giAreaMemSzReal = giAreaH * giAreaH * sizeof(hipfftReal);
const int	giScnOffset = giScnBegY * giScnW;
const int   giOrigScnMemSzUChar = giOrigScnSz * sizeof(unsigned char);

//directory where scene and templates are
char g_sPathBegin[50] = "../../cpuResults/";
char g_sPath[100];
//directory where stats files will be stored
char g_sStatsPathBegin[50] = "../stats/ssd_gpu_stats/fft_results/";
char g_sStatsPath[100];
FILE* g_fStatsFile;
//directory where scnbin files will be stored
#ifdef US_SIGNS
char g_sScnBinPathBegin[50] = "../convert_pgm_to_RawVideo/raw/";
#else
char g_sScnBinPathBegin[60] = "../../../copied15May17/EU_raw(savedRealisFilesAsBin)/";
#endif
char g_sScnBinPath[100];
FILE* g_fScnBin;





				//unsigned int guiParTim;
StopWatchInterface *guiParTim;
//unsigned int guiKerTim;
StopWatchInterface *guiKerTim;
double g_dRunsOnGPUTotalTime;
double g_dTotalKerTime;
double g_dClaheTime;

int giTplH, giTplW, giTplSz, giTplWMemSz, giTplMemSzReal, giTplMemSzCmplx;
int giNumIPRot, giNumSz, giNumOrigFlt, giNumSngCompFlt;

typedef struct CompFlt_struct
{
	float* h_afData;
	int iH;
	int iW;
	int iNumIPRot;
	int iNumSz;
	int iNumOrigFlt;
	int iNumMulCompFlt;
	int iDataSz;
	int iDataMemSz;
	int* aiIPAngs;
	int* aiTplCols;
	int* aiTpl_no;
}CompFlt_struct_t;

CompFlt_struct_t gstCompFlt;

int giPartMaxGDx, giWholeMaxGDx;
hipfftReal
*gd_pfMax,
*gd_afBlockMaxs;
int
*gd_piMaxIdx,
*gd_aiBlockMaxIdxs;
////////////////////////////////////////////////////////////////////////////////
// Following variables have been made global, so that we can divide the main function
// to init, fingBestTpl, and exit
////////////////////////////////////////////////////////////////////////////////


//typedef float hipfftReal;
hipfftReal
*gd_afScnPartIn,
*gh_afArea,
*gd_afCompFlt,
*gd_afPadTplIn,
*gd_afPadScnIn,
*gd_afCorr;

//typedef float hipfftComplex[2];
hipfftComplex
*gd_afScnPartOut,
*gd_afPadTplOut,
*gd_afPadScnOut,
*gd_afWholeTplFFT,
*gd_afPartTplFFT,
*gd_afMul;

unsigned char
*gh_acScn;

uchar4
*gd_ac4Scn;

hipfftHandle
ghFFTplanWholeFwd,
ghFFTplanWholeInv,
ghFFTplanPartFwd,
ghFFTplanPartInv;

dim3 gdThreadsConv(1, 1, 1);
dim3 gdBlocksConv(1, 1);
dim3 gdThreadsDead(1, 1, 1);
dim3 gdBlocksDead(1, 1);
dim3 gdThreadsWhole(1, 1, 1);
dim3 gdBlocksWhole(1, 1);
dim3 gdThreadsPart(1, 1, 1);
dim3 gdBlocksPart(1, 1);

int
giBegIdxIPInFirst,
giEndIdxIPInFirst,
giNumIPInFirst,
giBegIdxIPInSecond,
giEndIdxIPInSecond;

//adjust contrast and do gamma correction 
bool gbConGam = 0;
//fix the dead pixels in the given scene if we are processing a video 
bool gbFixDead = 1;

//params related to ConGam
#define LUTSIZE 256
float gfLUT[LUTSIZE];
unsigned char gacLUT[LUTSIZE];
float gfLIn = 0.2f;//0.4f;//0.2f;
float gfHIn = 0.8f;//0.6f;//0.8f;
float gfLOut = 0.0f;
float gfHOut = 1.0f;
float gfG = 2.5f;//0.5f;//2.5f;

				 //pass the found Speed Limit Number to the callee (GUI)
int giSLCurFrm = -1; //SL found in the current frame (-1 means no SL)
int giSLResult = -1; //SL found as a result of temporal integration (-1 means no SL)
int giShowClaheGUI = 0; //allow ssd_fft_GUI to turn on/off CLAHE showing (to capture the CLAHE effect in DAGM video) if -1 show, if 0 do not.
char gacClipName[11];
////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int iX, int iY) {
	return (iX % iY != 0) ? (iX / iY + 1) : (iX / iY);
}

//Align a to nearest higher multiple of b
int iAlignUp(int iX, int iY) {
	return (iX % iY != 0) ? (iX - iX % iY + iY) : iX;
}

//convert 1D Index to 2D Coordinates
void Indx2Coord(int iImgW, int iIndx, int* iRow, int* iCol)
{
	//cuda is row major and zero-based
	*iCol = iIndx%iImgW;
	//regular division of integer returns floor
	*iRow = iIndx / iImgW;
}

//convert 2D Coordinates to 2D Index
void Coord2Indx(int iImgW, int iRow, int iCol, int* iIndx)
{
	//cuda is row major and zero-based
	*iIndx = (iImgW*iRow) + iCol;
}

//assign values to rectangle specified by coord
void assignVal(int iImgW, float* afImg, int4 aiCoord, float fVal)
{
	int iIndx;
	for (int iRow = aiCoord.x; iRow <= aiCoord.y; iRow++)
	{
		for (int iCol = aiCoord.z; iCol <= aiCoord.w; iCol++)
		{
			Coord2Indx(iImgW, iRow, iCol, &iIndx);
			afImg[iIndx] = fVal;
		}
	}
}

//sum elements
float sum(float* afImg, int iSz)
{
	float fTotal = 0;
	for (int i = 0; i<iSz; i++)
	{
		fTotal += afImg[i];
	}
	return fTotal;
}

//get surrounding coordinates of the areas centered around a point 
int4 getSurrCoord(int iRow, int iCol, int iSurrH, int iNumCols, int iNumRows)
{
	//TODO: maybe I should shift area if it is at border, to produce lower PSR?
	int iHalfSurrH = iSurrH / 2;
	int iSurrRowBeg = iRow - iHalfSurrH + 1;
	if (iSurrRowBeg < 0) iSurrRowBeg = 0;
	int iSurrRowEnd = iRow + iHalfSurrH;
	if (iSurrRowEnd >= iNumRows) iSurrRowEnd = iNumRows - 1;
	int iSurrColBeg = iCol - iHalfSurrH + 1;
	if (iSurrColBeg < 0) iSurrColBeg = 0;
	int iSurrColEnd = iCol + iHalfSurrH;
	if (iSurrColEnd >= iNumCols) iSurrColEnd = iNumCols - 1;
	int4 aiAreaCoord = { iSurrRowBeg, iSurrRowEnd, iSurrColBeg, iSurrColEnd };

	return aiAreaCoord;
}

//make FFT size power of two 
int getPOTSz(int iSz) {
	//Highest non-zero bit position of iSz
	int iHiBit;
	//Neares lower and higher powers of two numbers for iSz
	unsigned int uiLowPOT, uiHiPOT;

	//Find highest non-zero bit (1U is unsigned one)
	for (iHiBit = 31; iHiBit >= 0; iHiBit--)
		if (iSz & (1U << iHiBit)) break;

	//No need to align, if already power of two
	uiLowPOT = 1U << iHiBit;
	if (uiLowPOT == iSz) return iSz;

	//Align to a nearest higher power of two, if the size is small enough,
	//else align only to a nearest higher multiple of 512,
	//in order to save computation and memory bandwidth
	uiHiPOT = 1U << (iHiBit + 1);
	if (uiHiPOT <= 1024)
		return uiHiPOT;
	else
		return iAlignUp(iSz, 512);
}


//Get the full path name
char* getFullPathOfFile(char* pcFileName)
{
	strcpy(g_sPath, g_sPathBegin);
	strcat(g_sPath, pcFileName);
	return g_sPath;
}

//compare GPU results to CPU results
void cmpCPU(void* afVals, char* pcFileName, bool bComplex, int iSz, bool bHost, float fEpsilon)
{
	int iMemSzReal = iSz * sizeof(hipfftReal);
	FILE* fRef = fopen(getFullPathOfFile(pcFileName), "rb");

	if (bComplex)
	{
		//file has both real and imaginary values
		int iMemSzCmplx = iSz * sizeof(hipfftComplex);
		hipfftComplex* h_afCmplx;
		if (!bHost)
		{
			h_afCmplx = (hipfftComplex *)malloc(iMemSzCmplx);
			// copy result from device to host
			CUDA_SAFE_CALL(hipMemcpy(h_afCmplx, (hipfftComplex*)afVals, iMemSzCmplx, hipMemcpyDeviceToHost));
		}
		else
		{
			h_afCmplx = (hipfftComplex *)afVals;
		}
		//extract real and imaginary parts
		float* h_afReal = (float*)malloc(iMemSzReal);
		float* h_afImag = (float*)malloc(iMemSzReal);
		for (int iI = 0; iI < iSz; iI++)
		{
			h_afReal[iI] = h_afCmplx[iI].x;
			h_afImag[iI] = h_afCmplx[iI].y;
		}

		// allocate mem to hold CPU results 
		float* afRealRef = (float*)malloc(iMemSzReal);
		float* afImagRef = (float*)malloc(iMemSzReal);
		fread(afRealRef, sizeof(float), iSz, fRef);
		fread(afImagRef, sizeof(float), iSz, fRef);

		CUTBoolean cutbResReal = cutComparefe(afRealRef, h_afReal, iSz, fEpsilon);
		CUTBoolean cutbResImag = cutComparefe(afImagRef, h_afImag, iSz, fEpsilon);
		printf("Checking %s result: %s\n", pcFileName, (1 == (cutbResReal && cutbResImag)) ? "PASSED" : "FAILED");


		//clean up memory
		if (!bHost) { free(h_afCmplx); }
		free(h_afReal);
		free(h_afImag);

		free(afRealRef);
		free(afImagRef);
	}
	else
	{
		//file has only real values
		int iMemSzReal = iSz * sizeof(hipfftReal);
		hipfftReal* h_afReal;
		if (!bHost)
		{
			h_afReal = (hipfftReal *)malloc(iMemSzReal);
			// copy result from device to host
			CUDA_SAFE_CALL(hipMemcpy(h_afReal, (hipfftReal*)afVals, iMemSzReal, hipMemcpyDeviceToHost));
		}
		else
		{
			h_afReal = (hipfftReal *)afVals;
		}

		// allocate mem to hold CPU results 
		float* afRealRef = (float*)malloc(iMemSzReal);
		fread(afRealRef, sizeof(float), iSz, fRef);

		CUTBoolean cutbResReal = cutComparefe(afRealRef, h_afReal, iSz, fEpsilon);
		printf("Checking %s result: %s\n", pcFileName, (cutbResReal) ? "PASSED" : "FAILED");


		//clean up memory
		if (!bHost) { free(h_afReal); }
		free(afRealRef);
	}
	fclose(fRef);

}

CompFlt_struct_t readCompFlt()
{
	CompFlt_struct_t gstCompFlt;
	FILE *fCompFlts = fopen(getFullPathOfFile("CompFlts.bin"), "rb");
	fread(&gstCompFlt.iH, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iW, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumIPRot, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumSz, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumOrigFlt, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumMulCompFlt, sizeof(int), 1, fCompFlts);
	int iNumTpl = gstCompFlt.iNumOrigFlt - gstCompFlt.iNumMulCompFlt;
	int iNumIPRotMemSz = gstCompFlt.iNumIPRot * sizeof(int);
	int iNumSzMemSz = gstCompFlt.iNumSz * sizeof(int);
	int iNumTplMemSz = iNumTpl * sizeof(int);
	int iNumAccResMemSz = iNumTpl * sizeof(AccRes_struct_t);
	gstCompFlt.iDataSz = gstCompFlt.iH * gstCompFlt.iW * gstCompFlt.iNumIPRot * gstCompFlt.iNumSz * gstCompFlt.iNumOrigFlt;
	gstCompFlt.iDataMemSz = gstCompFlt.iDataSz * sizeof(float);
#ifdef PINNED_MEM
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiIPAngs, iNumIPRotMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiTplCols, iNumSzMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiTpl_no, iNumTplMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.h_afData, gstCompFlt.iDataMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gastAccRes, iNumAccResMemSz));
#else
	gstCompFlt.aiIPAngs = (int *)malloc(iNumIPRotMemSz);
	gstCompFlt.aiTplCols = (int *)malloc(iNumSzMemSz);
	gstCompFlt.aiTpl_no = (int *)malloc(iNumTplMemSz);
	gstCompFlt.h_afData = (float *)malloc(gstCompFlt.iDataMemSz);
	gastAccRes = (float *)malloc(iNumAccResMemSz);
#endif
	fread(gstCompFlt.aiIPAngs, sizeof(int), gstCompFlt.iNumIPRot, fCompFlts);
	fread(gstCompFlt.aiTplCols, sizeof(int), gstCompFlt.iNumSz, fCompFlts);
	fread(gstCompFlt.aiTpl_no, sizeof(int), iNumTpl, fCompFlts);

	fread(gstCompFlt.h_afData, sizeof(float), gstCompFlt.iDataSz, fCompFlts);
	fclose(fCompFlts);
	//initialized the accpsr to zero
	memset(gastAccRes, '\0', iNumAccResMemSz);
	return gstCompFlt;
}
void getKernelDims(int iBlockDimX, int iSz, dim3* dThreads, dim3* dBlocks)
{
	(*dThreads).x = iBlockDimX;
	int iGDx = (iSz) % (iBlockDimX) > 0 ? ((iSz) / (iBlockDimX)) + 1 : (iSz) / (iBlockDimX);
	(*dBlocks).x = iGDx;
	return;
}


inline void InitKerTim(int iSz)
{
#ifdef KERTIM
	if (iSz == giTplSz)
	{
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUT_SAFE_CALL(cutResetTimer(guiKerTim));
		CUT_SAFE_CALL(cutStartTimer(guiKerTim));
	}
#endif
}

inline void WrapKerTim(char* sKerName, int iSz)
{
#ifdef KERTIM
	if (iSz == giTplSz) //1(copyscn convert fix), 2(1stPassInit), 3(2ndPassInit), giScnSz (1stLoop), giTplSz(2ndLoop)
	{
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUT_SAFE_CALL(cutStopTimer(guiKerTim));
		double dTime = sdkGetTimerValue(guiKerTim);
		printf("%s time: %f msecs.\n", sKerName, dTime);
		g_dTotalKerTime += dTime;
	}
#endif
}

void MaxIdx(hipfftReal* d_afData, int iSz, int** d_piMaxIdx)
{
	int iGDx;
	if (iSz == giScnSz)
		iGDx = giWholeMaxGDx; // = (640*480/512*8) = (307200/4096) = 75 - will need two passes 
	else
		iGDx = giPartMaxGDx;; // if TplSz = 60, (60*60/512*8)+1 = (3600/4096)+1 = 1 - will only need one pass 
							  //if TplSz is larger it is possible that we need two passes.
							  //gd_afBlockMaxs have enough storage for finding max in whole scene.
							  //so it is definitely enough for finding max in part scene

							  //max will do 2 passes. In the first pass there will be several blocks. 
							  //In the second	there will be only one block.

							  //now do the first pass: each thread will read EACHTHREADREADS pixels. 
							  //Each block reads BLOCKDIMX_MAX*EACHTHREADREADS = 512*8 = 4096 pixels

	dim3 thread(BLOCKDIMX_MAX, 1, 1);
	dim3 grid(iGDx, 1);
	
	//calculate block maxs
	InitKerTim(iSz);
	max_k << < grid, thread >> >(d_afData, NULL, iSz, gd_afBlockMaxs, gd_aiBlockMaxIdxs);
	WrapKerTim("Max1stPass", iSz);
	CUT_CHECK_ERROR("Kernel execution failed");

	if (iGDx == 1)
	{
		*d_piMaxIdx = gd_aiBlockMaxIdxs;
	}
	else
	{
		//now do the second pass: each thread will read EACHTHREADREADS blockmaxs. 
		//We have only one block and this block reads iGDx blockmaxs.
		//note that (iGDx/EACHTHREADREADS) <= BLOCKDIMX_MAX
		dim3 thread2(BLOCKDIMX_MAX, 1, 1);
		dim3 grid2(1, 1);
		
		// execute the kernel
		//calculate maxs of block maxs
		InitKerTim(iSz);
		max_k << < grid2, thread2 >> >(gd_afBlockMaxs, gd_aiBlockMaxIdxs, iGDx, gd_pfMax, gd_piMaxIdx);
		WrapKerTim("Max2ndPass", iSz);
		*d_piMaxIdx = gd_piMaxIdx;
		CUT_CHECK_ERROR("Kernel execution failed");
	}
}

//compute PSR value
float getPSR(hipfftReal* gd_afCorr, hipfftReal* gh_afArea, int* iPeakIndx, int iSz, int iW, int iH)
{
	int iI;
	int *d_piMaxIdx = NULL;
	MaxIdx(gd_afCorr, iSz, &d_piMaxIdx);
	InitKerTim(iSz);
	CUDA_SAFE_CALL(hipMemcpy(iPeakIndx, (int*)d_piMaxIdx, sizeof(int), hipMemcpyDeviceToHost));
	WrapKerTim("MemcpyD2HPeak", iSz);
	//find PSR on the cpu, because we are dealing with at most giAreaH x giAreaH elements
	int iMaxRow, iMaxCol;
	Indx2Coord(iW, *iPeakIndx, &iMaxRow, &iMaxCol);
	//The int4 type is a CUDA built-in type with four fields: x(RowBeg),y(RowEnd),z(ColBeg),w(ColEnd)
	int4 aiAreaCoord = getSurrCoord(iMaxRow, iMaxCol, giAreaH, iW, iH);
	int iStart = (aiAreaCoord.x*iW) + aiAreaCoord.z;
	//area is not always giAreaH x giAreaH, it might be cut if the peak is close to boundary
	int iNewAreaH = aiAreaCoord.y - aiAreaCoord.x + 1;
	int iNewAreaW = aiAreaCoord.w - aiAreaCoord.z + 1;
	int iNewAreaSz = iNewAreaW*iNewAreaH;
	//transfer the area
	InitKerTim(iSz);
	CUDA_SAFE_CALL(hipMemcpy2D(gh_afArea, iNewAreaW * sizeof(hipfftReal), gd_afCorr + iStart, iW * sizeof(hipfftReal), iNewAreaW * sizeof(hipfftReal), iNewAreaH, hipMemcpyDeviceToHost));
	WrapKerTim("MemcpyD2HArea", iSz);
	//find the new index of the max value in the area cut from corr plane
	float fMax = gh_afArea[0];
	int iNewMaxIndx = 0;
	for (iI = 0; iI<iNewAreaSz; iI++)
	{
		if (gh_afArea[iI] > fMax)
		{
			fMax = gh_afArea[iI];
			iNewMaxIndx = iI;
		}
	}
	int iNewMaxRow, iNewMaxCol;
	Indx2Coord(iNewAreaW, iNewMaxIndx, &iNewMaxRow, &iNewMaxCol);
	int4 aiMaskCoord = getSurrCoord(iNewMaxRow, iNewMaxCol, giMaskH, iNewAreaW, iNewAreaH);
	//mask is not always giMaskH x giMaskH, it might be cut if the peak is close to boundary
	int iNewMaskH = aiMaskCoord.y - aiMaskCoord.x + 1;
	int iNewMaskW = aiMaskCoord.w - aiMaskCoord.z + 1;
	//assign mask values to zero
	assignVal(iNewAreaW, gh_afArea, aiMaskCoord, 0);
	//calculate mean by not counting the mask
	int iFrameNumElem = (iNewAreaH*iNewAreaW) - (iNewMaskH*iNewMaskW);
	float fMean = sum(gh_afArea, iNewAreaSz) / iFrameNumElem;
	//mask values = mean
	assignVal(iNewAreaW, gh_afArea, aiMaskCoord, fMean);
	//calculate standard deviation by not counting the mask
	//calculate sum of sqr_dif
	float fTotal = 0;
	float fVal;
	for (iI = 0; iI < iNewAreaSz; iI++)
	{
		fVal = gh_afArea[iI] - fMean;
		fTotal += fVal*fVal;
	}
	float afStdVar = sqrt(fTotal / (iFrameNumElem - 1));
	float fMeasure;
	if (afStdVar != 0)
		fMeasure = (fMax - fMean) / afStdVar;
	else
		//if we are out of bound while copying part scene, this might happen since part scene will have lots of zeros
		fMeasure = 0;
	return fMeasure;
}

void Corr(hipfftComplex* d_afTplOut, dim3 dBlocks, dim3 dThreads, hipfftComplex* d_afScnOut, int iSz, hipfftComplex* gd_afMul, hipfftHandle hFFTplanInv, hipfftReal* gd_afCorr, hipfftReal* gh_afArea, int* piPeakIndx, float* pfPSR, int iW, int iH, bool szi)
{   
	//take conjugate of template fft and point wise multiply with scene and scale it with image size
	InitKerTim(iSz);
	pointWiseMul << <dBlocks, dThreads >> >(gd_afMul, d_afScnOut, d_afTplOut, iSz, 1.0f / (float)iSz);
	WrapKerTim("Mul", iSz);
	CUT_CHECK_ERROR("pointWiseMul() execution failed\n");
	//take inverse FFT of multiplication
	InitKerTim(iSz);
	CUFFT_SAFE_CALL(hipfftExecC2R(hFFTplanInv, (hipfftComplex *)gd_afMul, (hipfftReal *)gd_afCorr));
	WrapKerTim("FFTinv", iSz);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	//find the PSR
	*pfPSR = getPSR(gd_afCorr, gh_afArea, piPeakIndx, iSz, iW, iH);
	return;
}
inline void InitTim()
{
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkResetTimer(guiParTim));
	CUT_SAFE_CALL(sdkStartTimer(guiParTim));
#endif
}

inline void WrapTim(char* sParName)
{
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkStopTimer(guiParTim));
	double dTime = sdkGetTimerValue(guiParTim);
	printf("%s time: %f msecs.\n", sParName, dTime);
	g_dRunsOnGPUTotalTime += dTime;
#endif
}

void PrepTplFFT(hipfftReal* gd_afCompFlt, hipfftReal** d_pafPadTplIn, hipfftComplex** d_pafPadTplOut, hipfftComplex** d_pafWholeTplFFT, hipfftComplex** d_pafPartTplFFT, hipfftHandle ghFFTplanWholeFwd, hipfftHandle ghFFTplanPartFwd)
{
#ifdef SAVEFFT
	int iSzIndx, iIPIndx, iFltIndx, iFltAbsIndx, iIPAbsIndx;
	hipfftReal
		*d_afTpl,
		*d_afPadTplIn;
	//first allocate mem
	//WholeTpls are the MulCompFlts (last flts in the compflt list). They are used in 1st pass. Their size is as big as scn
	//PartTpls are all other comp flt excluding MulCompFlts. They are used in 2nd pass. Their size is as big as tpl (is not blowed up to scn size)
	int iWholeMemSz = giScnH * giScnW * giNumIPInFirst * giNumSz * gstCompFlt.iNumMulCompFlt * sizeof(hipfftComplex);
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafWholeTplFFT, iWholeMemSz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTplIn, giScnMemSzReal));
	int iPartMemSz = giTplH * giTplW * giNumIPRot * giNumSz * giNumSngCompFlt * sizeof(hipfftComplex);
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPartTplFFT, iPartMemSz));
	//take FFT of WholeTpls
	for (iFltIndx = giNumSngCompFlt; iFltIndx < giNumOrigFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = giBegIdxIPInFirst; iIPIndx < giEndIdxIPInFirst; iIPIndx++)
			{
				CUDA_SAFE_CALL(hipMemset(d_afPadTplIn, 0, giScnMemSzReal));
				d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
				//pad template
				CUDA_SAFE_CALL(hipMemcpy2D(d_afPadTplIn, (giScnW * sizeof(hipfftReal)), d_afTpl, giTplWMemSz, giTplWMemSz, giTplH, hipMemcpyDeviceToDevice));
				//take the fft and save it to WholeTplFFT
				iFltAbsIndx = iFltIndx - giNumSngCompFlt;
				iIPAbsIndx = iIPIndx - giBegIdxIPInFirst;
				//printf("iIPIndx=%d iSzIndx=%d iFltIndx=%d d_afPadTplIn= %d\n", iIPIndx, iSzIndx, iFltIndx, d_afPadTplIn);
				CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)d_afPadTplIn, (hipfftComplex *)*d_pafWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx)));

			}
		}
	}
	CUDA_SAFE_CALL(hipFree(d_afPadTplIn));
	//take FFT of PartTpls
	for (iFltIndx = 0; iFltIndx < giNumSngCompFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = 0; iIPIndx < giNumIPRot; iIPIndx++)
			{
				d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
				//printf("iIPIndx=%d iSzIndx=%d iFltIndx=%d d_afTpl= %d\n", iIPIndx, iSzIndx, iFltIndx, d_afTpl);
				//take the fft and save it to PartTplFFT
				CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)d_afTpl, (hipfftComplex *)*d_pafPartTplFFT(iIPIndx, iSzIndx, iFltIndx)));
			}
		}
	}
#else
	//allocate gd_afPadTplIn and gd_afPadTplOut 
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPadTplIn, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMemset(*d_pafPadTplIn, 0, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPadTplOut, giScnMemSzCmplx));
#endif
}

void DestroyTplFFT(hipfftComplex* gd_afWholeTplFFT, hipfftComplex* gd_afPartTplFFT, hipfftReal* gd_afPadTplIn, hipfftComplex* gd_afPadTplOut)
{
#ifdef SAVEFFT
	CUDA_SAFE_CALL(hipFree(gd_afWholeTplFFT));
	CUDA_SAFE_CALL(hipFree(gd_afPartTplFFT));
#else
	CUDA_SAFE_CALL(hipFree(gd_afPadTplIn));
	CUDA_SAFE_CALL(hipFree(gd_afPadTplOut));
#endif
}

void getWholeTplFFT(hipfftReal* gd_afCompFlt, int iIPIndx, int iSzIndx, int iFltIndx, hipfftReal* gd_afPadTplIn, hipfftComplex** d_pafPadTplOut, hipfftHandle ghFFTplanWholeFwd, hipfftComplex* gd_afWholeTplFFT)
{
#ifdef SAVEFFT
	int iFltAbsIndx = iFltIndx - giNumSngCompFlt;
	int iIPAbsIndx = iIPIndx - giBegIdxIPInFirst;
	*d_pafPadTplOut = gd_afWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx);
#else
	//find the starting index of template
	hipfftReal* d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
	//pad template
	CUDA_SAFE_CALL(hipMemcpy2D(gd_afPadTplIn, (giScnW * sizeof(hipfftReal)), d_afTpl, giTplWMemSz, giTplWMemSz, giTplH, hipMemcpyDeviceToDevice));
	//take the FFT of the template
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)gd_afPadTplIn, (hipfftComplex *)*d_pafPadTplOut));
#endif
}

void getPartTplFFT(hipfftReal* gd_afCompFlt, int iIPIndx, int iSzIndx, int iFltIndx, hipfftComplex** d_pafPadTplOut, hipfftHandle ghFFTplanPartFwd, hipfftComplex* gd_afPartTplFFT)
{
#ifdef SAVEFFT
	*d_pafPadTplOut = gd_afPartTplFFT(iIPIndx, iSzIndx, iFltIndx);
#else
	//get the pointer to the tpl
	hipfftReal* d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
	//take the FFT of the template
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)d_afTpl, (hipfftComplex *)*d_pafPadTplOut));
#endif
}

//If the MaxPeakIndex is close to the boundry of the image, when we try to copy the part of the scene
//we can be out bound! check for this condition, and if so get part of the scene until boundry.
//since we initalize the part image to zero, it would have an effect such that part image is padded with zero
void getCopyWidthHeight(int iMaxPeakIndx, int* piPartW, int* piPartH)
{
	int iMaxPeakRow, iMaxPeakCol;
	//make sure we are not out of bounds
	Indx2Coord(giScnW, iMaxPeakIndx, &iMaxPeakRow, &iMaxPeakCol);
	*piPartW = giTplW;
	int iEndCol = iMaxPeakCol + *piPartW - 1;
	if (iEndCol >= giScnW)
		*piPartW = *piPartW - (iEndCol + 1 - giScnW);
	*piPartH = giTplH;
	int iEndRow = iMaxPeakRow + *piPartH - 1;
	if (iEndRow >= giScnH)
		*piPartH = *piPartH - (iEndRow + 1 - giScnH);
}

/*B = GRAYTO8(A) converts the double array A to unisgned char by scaling A by 255
* and then rounding.  NaN's in A are converted to 0.  Values in A greater
* than 1.0 are converted to 255; values less than 0.0 are converted to 0.
*/
void ConvertFromDouble(float *pr, unsigned char *qr, int numElements)
{
	int k;
	float val;

	for (k = 0; k < numElements; k++)
	{
		val = *pr++;
		if (val == NULL) {
			*qr++ = 0;
		}
		else {
			val = val * 255.0f + 0.5f;
			if (val > 255.0) val = 255.0;
			if (val < 0.0)   val = 0.0;
			*qr++ = (unsigned char)val;
		}
	}
}

//this function immitates Matlab imadjust function's LookUp Table creation.
void genLUT()
{
	float fN = LUTSIZE;
	float fD1 = 0;
	float fD2 = 1;
	for (int i = 0; i < fN - 1; i++)
	{
		gfLUT[i] = fD1 + i*((fD2 - fD1) / (fN - 1));
	}
	gfLUT[int(fN - 1)] = fD2;

	//make sure lut is in the range [gfLIn;gfHIn]
	for (int i = 0; i < fN; i++)
	{
		if (gfLUT[i] < gfLIn) gfLUT[i] = gfLIn;
		if (gfLUT[i] > gfHIn) gfLUT[i] = gfHIn;
	}

	//out = ( (img - lIn(d,:)) ./ (hIn(d,:) - lIn(d,:)) ) .^ (g(d,:));
	for (int i = 0; i < fN; i++)
	{
		gfLUT[i] = pow((gfLUT[i] - gfLIn) / (gfHIn - gfLIn), gfG);
	}
	//out(:) = out .* (hOut(d,:) - lOut(d,:)) + lOut(d,:);
	for (int i = 0; i < fN; i++)
	{
		gfLUT[i] = gfLUT[i] * (gfHOut - gfLOut) + gfLOut;
	}
	ConvertFromDouble(gfLUT, gacLUT, LUTSIZE);
}

void CpyScnToDevAndPreProcess(unsigned char* acScn, float* d_afPadScnIn, bool bConGam, bool bFixDead)
{
	//I can do the adjusting before fixing the dead pixel. Adjusted dead pixel will be overwritten as an overage of adjusted neighbors. Adjusting is done to each pixel independently.
	//copy scene to device
	InitTim();
	InitKerTim(1);
	CUDA_SAFE_CALL(hipMemcpy(gd_ac4Scn, acScn + giScnOffset, giScnMemSzUChar, hipMemcpyHostToDevice));
	WrapKerTim("MemcpyH2DScn", 1);
	WrapTim("CopyFrameToGPUMem");

	InitTim();
	InitKerTim(1);
	convertChar4ToFloatDoConGam << <gdBlocksConv, gdThreadsConv >> > (gd_ac4Scn, (float4*)d_afPadScnIn, (giScnSz / 4), bConGam);
	WrapKerTim("ConvertScn", 1);
	WrapTim("convertChar4ToFloatDoConGam");

	if (bFixDead)
	{
		InitTim(); 
		InitKerTim(1);
		fixDeadPixels << <gdBlocksDead, gdThreadsDead >> > ((hipfftReal*)d_afPadScnIn, giScnSz, giScnW, giScnH);
		WrapKerTim("FixScn", 1);
		WrapTim("fixDeadPixel");
	}

#ifdef COPYBACKAFTERDEADFIX
	//only for visualization purposes. no need to optimize below code with kernels.
	/*hipfftReal* h_afScnOut = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScnOut, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	for (int i = 0; i < giScnSz; i++)
		acScn[i + giScnOffset] = (unsigned char)h_afScnOut[i];
	free(h_afScnOut);//????*/
#endif



#ifdef RUNCLAHE

	//IMPLEMENT THIS SECTION ON GPU: only for testing CLAHE it is running on the CPU
	hipfftReal* h_afScnClahe = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScnClahe, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	unsigned char* acScnClahe = (unsigned char*)malloc(giScnMemSzUChar);
	for (int i = 0; i < giScnSz; i++)
		{acScnClahe[i] = (unsigned char)h_afScnClahe[i];}
	//convert to unsigned int
	CLAHE(acScnClahe, giScnW, giScnH, 0, 255, giScnW / 8, giScnH / 8, 256, 0.3f); //80 60, 80 30
																				  //copy scene to device
	CUDA_SAFE_CALL(hipMemcpy(gd_ac4Scn, acScnClahe, giScnMemSzUChar, hipMemcpyHostToDevice));
	convertChar4ToFloatDoConGam << <gdBlocksConv, gdThreadsConv >> >(gd_ac4Scn, (float4*)d_afPadScnIn, (giScnSz / 4), bConGam);
	free(h_afScnClahe);
	//free(h_afScnOut);
	free(acScnClahe);

	////////////////////
#endif

	//added to show CLAHE effect in the ssd_fft_gpu_GUI
	if (giShowClaheGUI == -1)
	{
		//only for visualization purposes. no need to optimize below code with kernels.
		hipfftReal* h_afScnOutGUI = (hipfftReal*)malloc(giScnMemSzReal);
		CUDA_SAFE_CALL(hipMemcpy(h_afScnOutGUI, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
		for (int i = 0; i<giScnSz; i++)
			acScn[i + giScnOffset] = (unsigned char)h_afScnOutGUI[i];
		free(h_afScnOutGUI);
	}
}


void DisplayResults(float fPSR, int iTplIndx, int iIPIndx, int iSzIndx, int iStatsFrameCur)
{
	giSLCurFrm = -1;
	giSLResult = -1;
#ifdef DISP_FRM_RECOG
	if (fPSR > gfPSRTrashold)
	{
		//printf("Max PSR value: %f (TplNo = %d, IPAng = %d, Sz = %d)\n", fPSR, gstCompFlt.aiTpl_no[iTplIndx], gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
		printf("Frame votes for %3d %s (PSR: %5.2f, in-plane rotation: %3d\xf8, size: %2d)\n", gstCompFlt.aiTpl_no[iTplIndx], acMeasure, fPSR, gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
		giSLCurFrm = gstCompFlt.aiTpl_no[iTplIndx];
	}
	//	else
	//		printf("\n");
#endif

#ifdef MAJVOT
	int iNumTpl = gstCompFlt.iNumOrigFlt - gstCompFlt.iNumMulCompFlt;
	float fAddConfIP, fAddConfSz;

	//update the AccRes
	if (giFrameNo == 0)
	{
		if (fPSR > gfPSRTrashold)
		{
			//start the tracking at the first seen sign
			giFrameNo++;
			gastAccRes[iTplIndx].fAccConf = gastAccRes[iTplIndx].fAccConf + fPSR;
			giNumFramesInAcc++;
			gastAccRes[iTplIndx].iPrevIP = iIPIndx;
			gastAccRes[iTplIndx].iPrevSz = iSzIndx;
		}
	}
	else
	{
		//increase the tracked frameNum regardless of the PSR value if we already started the tracking
		giFrameNo++;
		if (fPSR > gfPSRTrashold)
		{
			fAddConfIP = 0;
			fAddConfSz = 0;
			if (gastAccRes[iTplIndx].fAccConf > 0)
			{
				//there has been a previous recognition of this tpl (iPrevIP and iPrevSz has valid values)
				//increase confidence if IP is the same as previous and/or Sz is getting larger.
				if ((iIPIndx - gastAccRes[iTplIndx].iPrevIP) == 0)
					fAddConfIP = gfAddConfIPFac*fPSR;
				if ((iSzIndx - gastAccRes[iTplIndx].iPrevSz) == 0)
					fAddConfSz = gfAddConfEqSzFac*fPSR;
				else if ((iSzIndx - gastAccRes[iTplIndx].iPrevSz) > 0)
					fAddConfSz = gfAddConfGrSzFac*fPSR;
			}
			gastAccRes[iTplIndx].fAccConf = gastAccRes[iTplIndx].fAccConf + fPSR + fAddConfIP + fAddConfSz;
			giNumFramesInAcc++;
			gastAccRes[iTplIndx].iPrevIP = iIPIndx;
			gastAccRes[iTplIndx].iPrevSz = iSzIndx;
		}
	}

	int iMaxTplIndx = -1;
	if (giFrameNo == giTrackingLen)
	{
		//find the bestTpl
		float fMaxAccConf = gastAccRes[0].fAccConf;
		iMaxTplIndx = 0;
		for (int i = 1; i<iNumTpl; i++)
		{
			if (gastAccRes[i].fAccConf > fMaxAccConf)
			{
				iMaxTplIndx = i;
				fMaxAccConf = gastAccRes[i].fAccConf;
			}
		}
		//printf("\n           Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
#ifdef REALTIME
		if (fMaxAccConf > gfAccPSRTrasholdSpecialReal && giNumFramesInAcc == 1 && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2)
			printf("\n           Best Tpl = %d (Max AccConf = %f)\n(special rule for realtime emulation=> result is based on only ONE frame with VERY high confidence)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
		else if (fMaxAccConf > gfAccPSRTrashold && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2) //2 = 00t
			printf("\n           Best Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
		else
			iMaxTplIndx = -1;
#else
		if (fMaxAccConf > gfAccPSRTrashold && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2) //2 = 00t
		{
			//printf("\n           Best Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
			printf("\n      System concludes that speed limit is %3d %s! (Total votes: %6.2f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], acMeasure, fMaxAccConf);
			giSLResult = gstCompFlt.aiTpl_no[iMaxTplIndx];
		}
		else
			iMaxTplIndx = -1;
#endif
		giFrameNo = 0;
		giNumFramesInAcc = 0;
		//initialize the accpsr to zero
		memset(gastAccRes, '\0', (iNumTpl * sizeof(AccRes_struct_t)));
	}
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Member Functions
////////////////////////////////////////////////////////////////////////////////
void ssd_fft_gpu_init()
{

	//read comp filters
	gstCompFlt = readCompFlt();//can't read
	giTplH = gstCompFlt.iH;//
	giTplW = gstCompFlt.iW;//
	giTplSz = giTplH * giTplW;
	giTplWMemSz = giTplW * sizeof(hipfftReal);
	giTplMemSzReal = giTplH * giTplW * sizeof(hipfftReal);
	giTplMemSzCmplx = giTplH * giTplW * sizeof(hipfftComplex);
	giNumIPRot = gstCompFlt.iNumIPRot;
	giNumSz = gstCompFlt.iNumSz;
	giNumOrigFlt = gstCompFlt.iNumOrigFlt;
	giNumSngCompFlt = giNumOrigFlt - gstCompFlt.iNumMulCompFlt;

	
	//do some check
	giPartMaxGDx = (giTplSz) % (BLOCKDIMX_MAX*EACHTHREADREADS) > 0 ? ((giTplSz) / (BLOCKDIMX_MAX*EACHTHREADREADS)) + 1 : (giTplSz) / (BLOCKDIMX_MAX*EACHTHREADREADS);
	if (giPartMaxGDx > 1)
	{
		printf("Warning: Max of part scn can not be found in one pass!\n");
	}
	giWholeMaxGDx = (giScnSz) % (BLOCKDIMX_MAX*EACHTHREADREADS) > 0 ? ((giScnSz) / (BLOCKDIMX_MAX*EACHTHREADREADS)) + 1 : (giScnSz) / (BLOCKDIMX_MAX*EACHTHREADREADS);
	if ((giWholeMaxGDx / EACHTHREADREADS) > BLOCKDIMX_MAX)
	{
		//in the second pass each thread will read EACHTHREADREADS blockmaxs. There is giWholeMaxGDx blocks at most.
		//if giWholeMaxGDx/EACHTHREADREADS > BLOCKDIMX_MAX this means that second pass should have more than one block.
		//but it should have only one!
		printf("Error: Each thread in max kernel should read more than %d elements!\n", EACHTHREADREADS);
		exit(0);
	}

#ifdef PINNED_MEM
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gh_acScn, giOrigScnMemSzUChar));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gh_afArea, giAreaMemSzReal));
#else
	gh_acScn = (unsigned char *)malloc(giOrigScnMemSzUChar);
	gh_afArea = (hipfftReal *)malloc(giAreaMemSzReal);
#endif
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_ac4Scn, giScnMemSzUChar));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afCompFlt, gstCompFlt.iDataMemSz));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnIn, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afCorr, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnOut, giScnMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afMul, giScnMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afScnPartIn, giTplMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afScnPartOut, giTplMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_pfMax, sizeof(hipfftReal)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_afBlockMaxs, sizeof(hipfftReal)*giWholeMaxGDx));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_piMaxIdx, sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_aiBlockMaxIdxs, sizeof(int)*giWholeMaxGDx));

	gd_afWholeTplFFT = NULL;
	gd_afPartTplFFT = NULL;
	gd_afPadTplIn = NULL;
	gd_afPadTplOut = NULL;


	//calculate the block and grid size (both will be 1D) to be used by kernels
	getKernelDims(BLOCKDIMX, giScnSz / 4, &gdThreadsConv, &gdBlocksConv);
	getKernelDims(BLOCKDIMX, giScnSz / 2, &gdThreadsDead, &gdBlocksDead);
	gdThreadsDead.x = gdThreadsDead.x + (HALFWARP + 1);
	getKernelDims(BLOCKDIMX, giScnSz, &gdThreadsWhole, &gdBlocksWhole);
	getKernelDims(BLOCKDIMX, giTplSz, &gdThreadsPart, &gdBlocksPart);

	//Creating FFT plan for whole scene
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanWholeFwd, giScnH, giScnW, HIPFFT_R2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanWholeInv, giScnH, giScnW, HIPFFT_C2R));
	//Creating FFT plan for part of the scene
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanPartFwd, giTplH, giTplW, HIPFFT_R2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanPartInv, giTplH, giTplW, HIPFFT_C2R));

	sdkCreateTimer(&guiParTim);
	sdkCreateTimer(&guiKerTim);

	InitTim();
	//copy all Composite Filters to device memory (copying device to device would take less time)
	CUDA_SAFE_CALL(hipMemcpy(gd_afCompFlt, gstCompFlt.h_afData, gstCompFlt.iDataMemSz, hipMemcpyHostToDevice));
	//figure out params regarding IPRot
#ifdef DoIPInSecond
	giBegIdxIPInFirst = giNumIPRot / 2; //middle is the not-IProtated compFlt
	giEndIdxIPInFirst = giBegIdxIPInFirst + 1;
	giNumIPInFirst = 1;
	giBegIdxIPInSecond = 0;
	giEndIdxIPInSecond = giNumIPRot;
#else
	giBegIdxIPInFirst = 0;
	giEndIdxIPInFirst = giNumIPRot;
	giNumIPInFirst = giNumIPRot;
	//assign second pass params on-line
#endif
	PrepTplFFT(gd_afCompFlt, &gd_afPadTplIn, &gd_afPadTplOut, &gd_afWholeTplFFT, &gd_afPartTplFFT, ghFFTplanWholeFwd, ghFFTplanPartFwd);
	WrapTim("PrepTplFFT");
	if (gbConGam)
	{
		genLUT();
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_acLUT), gacLUT, sizeof(unsigned char)*LUTSIZE));
	}

#ifndef REALTIME
	gfAccPSRTrashold = 31.5f;//32;//before: gfAccPSRFac*gfPSRTrashold if clahelimit = 0.6f fac = 4, if clahelimit= 0.3f (less noise) fac = 5 to avoid FP, TN
#else
	gfAccPSRTrashold = 25.95f;
#endif
}

void BestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp)
{
	int iIPIndx, iSzIndx, iFltIndx, iPeakIndx;

	float fPSR, fMaxPSR;

	int iMaxIPIndx, iMaxSzIndx, iMaxFltIndx;

	int iPartWMemSz;

	giShowClaheGUI = file_info[2];
	//save the scn in bin file (to transfer the videos from Realis to GUI)

	*piMaxPeakIndx = -1;
	*piPartW = -1;
	*piPartH = -1;


	bool bLoadScn = false;
	//Read scene...
	if (acScn == NULL)
	{
		//no video input, process the scn from file
		FILE *fScn = fopen(getFullPathOfFile("scn.bin"), "rb");
		fread(gh_acScn, sizeof(unsigned char), giOrigScnSz, fScn);
		fclose(fScn);
		acScn = gh_acScn;
		bLoadScn = true;
	}
	bool bFixDead = gbFixDead;
	if (bLoadScn) bFixDead = 0;

	////////FIRST PASS///////////

	CpyScnToDevAndPreProcess(acScn, gd_afPadScnIn, gbConGam, bFixDead);

	//Running the correlation...
	InitTim();
	//take the FFT of the scene
	InitKerTim(2);
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)gd_afPadScnIn, (hipfftComplex *)gd_afPadScnOut));
	WrapKerTim("wholeFFT", 2);
	//apply kth law to scene
	InitKerTim(2);
	kthLaw << <gdBlocksWhole, gdThreadsWhole >> >(gd_afPadScnOut, giScnSz);
	WrapKerTim("wholeKth", 2);
	//initialize max PSR value
	fMaxPSR = INT_MIN;
	//First find the peak with MulCompFlts
	WrapTim("FirstPassInit");
	InitTim();
	for (iFltIndx = giNumSngCompFlt; iFltIndx < giNumOrigFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = giBegIdxIPInFirst; iIPIndx < giEndIdxIPInFirst; iIPIndx++)
			{
				//I am not initializing gh_afArea. make sure you reach right coords.
				getWholeTplFFT(gd_afCompFlt, iIPIndx, iSzIndx, iFltIndx, gd_afPadTplIn, &gd_afPadTplOut, ghFFTplanWholeFwd, gd_afWholeTplFFT);
				//perform correlation
				Corr(gd_afPadTplOut, gdBlocksWhole, gdThreadsWhole, gd_afPadScnOut, giScnSz, gd_afMul, ghFFTplanWholeInv, gd_afCorr, gh_afArea, &iPeakIndx, &fPSR, giScnW, giScnH, false);
				//printf("PSR value for MulCompFlt: %f (iFltIndx = %d IPAng = %d, Sz = %d)\n", fPSR, iFltIndx, gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
				if (fPSR > fMaxPSR)
				{
					fMaxPSR = fPSR;
					iMaxIPIndx = iIPIndx;
					iMaxSzIndx = iSzIndx;
					*piMaxPeakIndx = iPeakIndx;
				}
			}
		}
	}
#ifndef DoIPInSecond
	giBegIdxIPInSecond = iMaxIPIndx;
	giEndIdxIPInSecond = giBegIdxIPInSecond + 1;
#endif
	WrapTim("FirstPassLoop");

	////////SECOND PASS///////////
	InitTim();
	//we know the max IP and Sz. Now try different templates
	//copy template-size portion of the scene starting at peak point
	//	CUDA_SAFE_CALL( hipMemcpy2D( gd_afScnPartIn, giTplWMemSz, gd_afPadScnIn+iMaxPeakIndx, giScnW*sizeof(hipfftReal), giTplWMemSz, giTplH , hipMemcpyDeviceToDevice ));
	getCopyWidthHeight(*piMaxPeakIndx, piPartW, piPartH);
	iPartWMemSz = *piPartW * sizeof(hipfftReal);
	//make sure you initialize gd_afScnPartIn with zeros before processing each frame (if we are out of bounds, we will have a part image padded with zeros)
	InitKerTim(3);
	CUDA_SAFE_CALL(hipMemset(gd_afScnPartIn, 0, giTplMemSzReal));
	CUDA_SAFE_CALL(hipMemcpy2D(gd_afScnPartIn, giTplWMemSz, gd_afPadScnIn + *piMaxPeakIndx, giScnW * sizeof(hipfftReal), iPartWMemSz, *piPartH, hipMemcpyDeviceToDevice));
	WrapKerTim("MemcpyD2DPart", 3);
	//take the FFT of the scene
	InitKerTim(3);
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)gd_afScnPartIn, (hipfftComplex *)gd_afScnPartOut));
	WrapKerTim("partFFT", 3);
	//apply kth law to scene
	InitKerTim(3);
	kthLaw << <gdBlocksPart, gdThreadsPart >> >(gd_afScnPartOut, giTplSz);
	WrapKerTim("partKth", 3);
	fMaxPSR = INT_MIN;
	WrapTim("SecondPassInit");
	InitTim();
	for (iFltIndx = 0; iFltIndx < giNumSngCompFlt; iFltIndx++)
	{
		for (iIPIndx = giBegIdxIPInSecond; iIPIndx < giEndIdxIPInSecond; iIPIndx++)
		{
			getPartTplFFT(gd_afCompFlt, iIPIndx, iMaxSzIndx, iFltIndx, &gd_afPadTplOut, ghFFTplanPartFwd, gd_afPartTplFFT);
			Corr(gd_afPadTplOut, gdBlocksPart, gdThreadsPart, gd_afScnPartOut, giTplSz, gd_afMul, ghFFTplanPartInv, gd_afCorr, gh_afArea, &iPeakIndx, &fPSR, giTplW, giTplH, true);
			if (fPSR > fMaxPSR)
			{
				fMaxPSR = fPSR;
				iMaxFltIndx = iFltIndx;
				iMaxIPIndx = iIPIndx;
			}
		}
	}
	WrapTim("SecondPassLoop");


	DisplayResults(fMaxPSR, iMaxFltIndx, iMaxIPIndx, iMaxSzIndx, file_info[0]);
	//in realis show the peak in correct position (add offset of the window in the frame)
#ifdef SHOWBOX_WHENRECOG
	if (fMaxPSR <= gfPSRTrashold) //hide the box upper left corner, if the PSR is below trashold
		*piMaxPeakIndx = 0 - ((giTplH*giScnW * 2) + giScnOffset);
#endif
	*piMaxPeakIndx = *piMaxPeakIndx + giScnOffset;
	//printf("MaxPeakIndx: %d, FrameID: %d\n", *piMaxPeakIndx, file_info[0]);
}


void ssd_fft_gpu_findBestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp)
{
	BestTpl(acScn, piMaxPeakIndx, piPartW, piPartH, file_info, ulTimeStamp);
}


void ssd_fft_gpu_returnBestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp, int* iSLCurFrm, int* iSLResult, char* acClipName)
{
	strcpy(gacClipName, acClipName);
	BestTpl(acScn, piMaxPeakIndx, piPartW, piPartH, file_info, ulTimeStamp);
	*iSLCurFrm = giSLCurFrm;
	*iSLResult = giSLResult;
}


void ssd_fft_gpu_exit() {


	printf("Shutting down...\n");
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanWholeFwd));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanWholeInv));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanPartFwd));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanPartInv));
	CUDA_SAFE_CALL(hipFree(gd_ac4Scn));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnIn));
	CUDA_SAFE_CALL(hipFree(gd_afScnPartIn));
	CUDA_SAFE_CALL(hipFree(gd_afScnPartOut));
	CUDA_SAFE_CALL(hipFree(gd_afCompFlt));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnOut));
	CUDA_SAFE_CALL(hipFree(gd_afCorr));
	CUDA_SAFE_CALL(hipFree(gd_afMul));
	CUDA_SAFE_CALL(hipFree(gd_pfMax));
	CUDA_SAFE_CALL(hipFree(gd_afBlockMaxs));
	CUDA_SAFE_CALL(hipFree(gd_piMaxIdx));
	CUDA_SAFE_CALL(hipFree(gd_aiBlockMaxIdxs));
	DestroyTplFFT(gd_afWholeTplFFT, gd_afPartTplFFT, gd_afPadTplIn, gd_afPadTplOut);
#ifdef PINNED_MEM
	hipHostFree(gh_acScn);
	hipHostFree(gh_afArea);
	hipHostFree(gstCompFlt.aiIPAngs);
	hipHostFree(gstCompFlt.aiTplCols);
	hipHostFree(gstCompFlt.aiTpl_no);
	hipHostFree(gstCompFlt.h_afData);
	hipHostFree(gastAccRes);
#else
	free(gh_acScn);
	free(gh_afArea);
	free(gstCompFlt.aiIPAngs);
	free(gstCompFlt.aiTplCols);
	free(gstCompFlt.aiTpl_no);
	free(gstCompFlt.h_afData);
	free(gastAccRes);
#endif
	//CUT_EXIT(argc, argv);
}

